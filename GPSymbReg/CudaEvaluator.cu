#include "hip/hip_runtime.h"
//
// Created by zac on 11.04.18..
//

#include "CudaEvaluator.h"

CudaEvaluator::CudaEvaluator(int N, int DIM, int MAX_PROG_SIZE, vector<vector<double>> &input) :
        N(N), DIM(DIM), MAX_PROG_SIZE(MAX_PROG_SIZE) {
    hipMalloc((void **) &d_program, MAX_PROG_SIZE * sizeof(uint));
    hipMalloc((void **) &d_programConst, MAX_PROG_SIZE * sizeof(double));
    hipMalloc((void **) &d_input, N * DIM * sizeof(double));
    hipMalloc((void **) &d_output, N * sizeof(double));
    hipMalloc((void **) &d_stack, N * MAX_PROG_SIZE * sizeof(double));

    //  copy input matrix to 1D array
    double *h_input = new double[N * DIM];
    double *p_input = h_input;
    for (int i = 0; i < N; i++) {
        copy(input[i].begin(), input[i].end(), p_input);
        p_input += DIM;
    }

    hipMemcpy(d_input, h_input, N * DIM * sizeof(double), hipMemcpyHostToDevice);

    delete[] h_input;

}

CudaEvaluator::~CudaEvaluator() {
    hipFree(d_program);
    hipFree(d_programConst);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_stack);
}

void CudaEvaluator::evaluate(vector<uint> &program, vector<double> &programConst,
                             vector<vector<double>> &input,
                             vector<double> &result) {


    int PROG_SIZE = program.size();

    hipMemcpy(d_program, &program[0], program.size() * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(d_programConst, &programConst[0], program.size() * sizeof(double), hipMemcpyHostToDevice);

    dim3 dimGridN(N, 1);
    dim3 dimBlock(1, 1, 1);

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    evaluateParallel<<<dimGridN, dimBlock>>>(d_program, d_programConst, d_input, d_output, d_stack, N, DIM, program.size());
    hipDeviceSynchronize();

    double *h_output = new double[N];
    hipMemcpy(h_output, d_output, N * sizeof(double), hipMemcpyDeviceToHost);

    std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
    std::cout << "GPU Time difference [us] = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() <<std::endl;

    result.resize(N, 0.);
    for (int i = 0; i < N; i++) {
        result[i] = h_output[i];
//        cout << h_output[i] << endl;
    }

    delete[] h_output;

}

__global__ void evaluateParallel(uint *d_program,
                                 double *d_programConstant,
                                 double *d_input,
                                 double *d_output,
                                 double *d_stack,
                                 int N, int DIM, int prog_size) {
    int tid = blockIdx.x;

    double *stack = d_stack + tid * prog_size;

    double *input = d_input + tid * DIM;

//    for (int i=0; i<prog_size; i++) {
//        t_stack[i] = (double) i;
//    }

    int SP = 0;

    double o1, o2, tmp;

    for (int i = 0; i < prog_size; i++) {
        switch (d_program[i]) {
            case ADD:
                o2 = stack[--SP];
                o1 = stack[--SP];

                tmp = o1 + o2;

                stack[SP++] = tmp;
                break;
            case SUB:
                o2 = stack[--SP];
                o1 = stack[--SP];

                tmp = o1 - o2;

                stack[SP++] = tmp;
                break;
            case MUL:
                o2 = stack[--SP];
                o1 = stack[--SP];

                tmp = o1 * o2;

                stack[SP++] = tmp;
                break;
            case DIV:
                o2 = stack[--SP];
                o1 = stack[--SP];

                tmp = (fabs(o2) > 0.000000001) ? o1 / o2 : 1.;

                stack[SP++] = tmp;
                break;
            case SQR:
                o1 = stack[--SP];

                tmp = (o1 >= 0.) ? sqrt(o1) : 1.;

                stack[SP++] = tmp;
                break;
            case SIN:
                o1 = stack[--SP];

                tmp = sin(o1);

                stack[SP++] = tmp;
                break;
            case COS:
                o1 = stack[--SP];

                tmp = cos(o1);

                stack[SP++] = tmp;
                break;
            case VAR_X0:
                tmp = input[0];

                stack[SP++] = tmp;
                break;
            case VAR_X1:
                tmp = input[1];

                stack[SP++] = tmp;
                break;
            case VAR_X2:
                tmp = input[2];

                stack[SP++] = tmp;
                break;
            case VAR_X3:
                tmp = input[3];

                stack[SP++] = tmp;
                break;
            case VAR_X4:
                tmp = input[4];

                stack[SP++] = tmp;
                break;
            case CONST:
                tmp = d_programConstant[i];

                stack[SP++] = tmp;
                break;
            case ERR:
            default:
//                cerr<< "ERRRRRRRRRRRRRRRRRRRRRRRRRRRRRRRRR" << endl;
                d_output[tid] = -1;
                return;
        }
    }

//    cerr << "SP:\t" << SP << endl;
    double result = stack[--SP];

    d_output[tid] = result;

}