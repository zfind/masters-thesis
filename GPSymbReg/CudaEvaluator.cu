#include "hip/hip_runtime.h"
#include <cmath>
//#include <ecf/ECF.h>
#include <stack>
#include <chrono>
#include <limits>

#include "CudaEvaluator.h"
#include "Constants.h"


#define EVALUATE_ERROR do {cerr << "ERRRRRRRRRRRRRRRRRRRRRRRRRRRRRRRRR" << endl; return NAN; } while(0);
#define GPU_EVALUATE_ERROR do {d_output[tid] = NAN; return;} while(0);




CudaEvaluator::CudaEvaluator(int N, int DIM, int MAX_PROG_SIZE, vector<vector<double>> &input, vector<double> &output) :
        N(N), DIM(DIM), MAX_PROG_SIZE(MAX_PROG_SIZE), datasetInput(input), datasetOutput(output) {
    hipMalloc((void **) &d_program, MAX_PROG_SIZE * sizeof(uint));
    hipMalloc((void **) &d_programConst, MAX_PROG_SIZE * sizeof(double));
    hipMalloc((void **) &d_input, N * DIM * sizeof(double));
    hipMalloc((void **) &d_output, N * sizeof(double));
    hipMalloc((void **) &d_stack, N * MAX_PROG_SIZE * sizeof(double));
    hipMalloc((void **) &d_real, N * sizeof(double));
    hipMalloc((void **) &d_answer, sizeof(double));

    //  copy input matrix to 1D array
    double *h_input = new double[N * DIM];
    double *p_input = h_input;
    for (int i = 0; i < N; i++) {
        copy(input[i].begin(), input[i].end(), p_input);
        p_input += DIM;
    }

    hipMemcpy(d_input, h_input, N * DIM * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_real, &datasetOutput[0], N * sizeof(double), hipMemcpyHostToDevice);

    hipHostMalloc((void**)&postfixMemPinned, MAX_PROG_SIZE*(sizeof(uint) + sizeof(double)));

    delete[] h_input;


    cerr << "INICIJALIZIRAM EVALUATOR" << endl;
}

CudaEvaluator::~CudaEvaluator() {
    hipFree(d_program);
    hipFree(d_programConst);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_stack);
}

double CudaEvaluator::d_evaluate(char* postfixMem, uint PROG_SIZE, uint CONST_SIZE,
                                 vector<double> &result) {


//    int PROG_SIZE = program.size();

    hipMemcpy(d_program, postfixMem, PROG_SIZE * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(d_programConst, postfixMem + PROG_SIZE * sizeof(uint), CONST_SIZE * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(128, 1);
    dim3 grid((N + block.x - 1) / block.x, 1);
    size_t shared_size = block.x * PROG_SIZE * sizeof(double);

//    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

//    double fitness;

    d_evaluateIndividualNew<<<grid, block, shared_size>>>(d_program, d_programConst,
            d_input, d_output, d_stack, d_real,
            N, DIM, PROG_SIZE, d_answer);
//    hipDeviceSynchronize();

//    hipMemcpy(&fitness, d_answer, sizeof(double), hipMemcpyDeviceToHost);

    result.resize(N, 0.);
//    double *h_output = new double[N];
    hipMemcpy(&result[0], d_output, N * sizeof(double), hipMemcpyDeviceToHost);

//    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
//    std::cout << "GPU Time difference [us] = "
//              << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << std::endl;

    double fitness = 0.;
    for (int i = 0; i < N; i++) {
        fitness += fabs(datasetOutput[i] - result[i]);
    }

    return fitness;
}



__global__ void d_evaluateIndividualNew(uint *d_program,
                                     double *d_programConstant,
                                     double *d_input,
                                     double *d_output,
                                     double *d_stack,
                                        double *d_real,
                                     int N, int DIM, int prog_size, double *d_answer) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= N) return;

//    double *stack = d_stack + tid * prog_size;

//    double stack[50];

    extern __shared__ double stackChunk[];
    double *stack = stackChunk + threadIdx.x * prog_size;

    double *input = d_input + tid * DIM;


    int SP = 0;

    double o1, o2, tmp;

    for (int i = 0; i < prog_size; i++) {
        if (d_program[i] >= ARR_2) {
            o2 = stack[--SP];
            o1 = stack[--SP];

            switch (d_program[i]) {
                case ADD:
                    tmp = o1 + o2;
                    break;
                case SUB:
                    tmp = o1 - o2;
                    break;
                case MUL:
                    tmp = o1 * o2;
                    break;
                case DIV:
                    tmp = (fabs(o2) > 0.000000001) ? o1 / o2 : 1.;
                    break;
                default:
                    GPU_EVALUATE_ERROR
            }


        } else if (d_program[i] >= ARR_1) {
            o1 = stack[--SP];

            switch (d_program[i]) {
                case SQR:
                    tmp = (o1 >= 0.) ? sqrt(o1) : 1.;
                    break;
                case SIN:
                    tmp = sin(o1);
                    break;
                case COS:
                    tmp = cos(o1);
                    break;
                default:
                    GPU_EVALUATE_ERROR
            }


        } else if (d_program[i] == CONST) {
            tmp = *d_programConstant;
            d_programConstant++;

        } else if (d_program[i] >= VAR && d_program[i] < CONST) {
            uint code = d_program[i];
            uint idx = code - VAR;
            tmp = input[idx];

        } else {
            GPU_EVALUATE_ERROR
        }

        stack[SP++] = tmp;
    }

    double result = stack[--SP];

    d_output[tid] = result;

//    __syncthreads();
//
//    if (tid == 0) {
//        result=0.;
//        #pragma unroll
//        for (uint i = 0; i < N; i++) {
//            result += fabs(d_real[i] - d_output[i]);
//        }
//        *d_answer = result;
//    }
}

__global__ void d_evaluateIndividual(uint *d_program,
                                     double *d_programConstant,
                                     double *d_input,
                                     double *d_output,
                                     double *d_stack,
                                     int N, int DIM, int prog_size) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= N) return;

//    double *stack = d_stack + tid * prog_size;

//    double stack[50];

    extern __shared__ double stackChunk[];
    double *stack = stackChunk + threadIdx.x * prog_size;

    double *input = d_input + tid * DIM;


    int SP = 0;

    double o1, o2, tmp;

    for (int i = 0; i < prog_size; i++) {
        if (d_program[i] >= ARR_2) {
            o2 = stack[--SP];
            o1 = stack[--SP];

            switch (d_program[i]) {
                case ADD:
                    tmp = o1 + o2;
                    break;
                case SUB:
                    tmp = o1 - o2;
                    break;
                case MUL:
                    tmp = o1 * o2;
                    break;
                case DIV:
                    tmp = (fabs(o2) > 0.000000001) ? o1 / o2 : 1.;
                    break;
                default:
                    GPU_EVALUATE_ERROR
            }


        } else if (d_program[i] >= ARR_1) {
            o1 = stack[--SP];

            switch (d_program[i]) {
                case SQR:
                    tmp = (o1 >= 0.) ? sqrt(o1) : 1.;
                    break;
                case SIN:
                    tmp = sin(o1);
                    break;
                case COS:
                    tmp = cos(o1);
                    break;
                default:
                    GPU_EVALUATE_ERROR
            }


        } else if (d_program[i] == CONST) {
            tmp = d_programConstant[i];

        } else if (d_program[i] >= VAR && d_program[i] < CONST) {
            uint code = d_program[i];
            uint idx = code - VAR;
            tmp = input[idx];

        } else {
            GPU_EVALUATE_ERROR
        }

        stack[SP++] = tmp;
    }

    double result = stack[--SP];

    d_output[tid] = result;
}


double CudaEvaluator::h_evaluateIndividual(std::vector<uint> &solution, std::vector<double> &solutionConst,
                                           std::vector<double> &input, int validLength) {
    double stack[validLength];
    int SP = 0;

    double o1, o2, tmp;

    for (int i = 0; i < validLength; i++) {
        if (solution[i] >= ARR_2) {
            o2 = stack[--SP];
            o1 = stack[--SP];

            switch (solution[i]) {
                case ADD:
                    tmp = o1 + o2;
                    break;
                case SUB:
                    tmp = o1 - o2;
                    break;
                case MUL:
                    tmp = o1 * o2;
                    break;
                case DIV:
                    tmp = (fabs(o2) > 0.000000001) ? o1 / o2 : 1.;
                    break;
                default:
                    EVALUATE_ERROR
            }


        } else if (solution[i] >= ARR_1) {
            o1 = stack[--SP];

            switch (solution[i]) {
                case SQR:
                    tmp = (o1 >= 0.) ? sqrt(o1) : 1.;
                    break;
                case SIN:
                    tmp = sin(o1);
                    break;
                case COS:
                    tmp = cos(o1);
                    break;
                default:
                    EVALUATE_ERROR
            }


        } else if (solution[i] == CONST) {
            tmp = solutionConst[i];

        } else if (solution[i] >= VAR && solution[i] < CONST) {
            uint code = solution[i];
            uint idx = code - VAR;
            tmp = input[idx];

        } else {
            EVALUATE_ERROR
        }

        stack[SP++] = tmp;
    }

    double result = stack[--SP];

    return result;
}

double CudaEvaluator::h_evaluateIndividualNew(char* postfixMem, uint PROG_SIZE, uint MEM_SIZE,
                                              std::vector<double> &input) {

    uint* program = (uint*) postfixMem;
    double* programConst = (double*) &program[PROG_SIZE];

    double stack[PROG_SIZE];
    int SP = 0;

    double o1, o2, tmp;

    for (int i = 0; i < PROG_SIZE; i++) {
        if (program[i] >= ARR_2) {
            o2 = stack[--SP];
            o1 = stack[--SP];

            switch (program[i]) {
                case ADD:
                    tmp = o1 + o2;
                    break;
                case SUB:
                    tmp = o1 - o2;
                    break;
                case MUL:
                    tmp = o1 * o2;
                    break;
                case DIV:
                    tmp = (fabs(o2) > 0.000000001) ? o1 / o2 : 1.;
                    break;
                default:
                    EVALUATE_ERROR
            }


        } else if (program[i] >= ARR_1) {
            o1 = stack[--SP];

            switch (program[i]) {
                case SQR:
                    tmp = (o1 >= 0.) ? sqrt(o1) : 1.;
                    break;
                case SIN:
                    tmp = sin(o1);
                    break;
                case COS:
                    tmp = cos(o1);
                    break;
                default:
                    EVALUATE_ERROR
            }


        } else if (program[i] == CONST) {
            tmp = *programConst;
            programConst++;

        } else if (program[i] >= VAR && program[i] < CONST) {
            uint code = program[i];
            uint idx = code - VAR;
            tmp = input[idx];

        } else {
            EVALUATE_ERROR
        }

        stack[SP++] = tmp;
    }

    double result = stack[--SP];

    return result;
}


double CudaEvaluator::h_evaluate(std::vector<uint> &program, std::vector<double> &programConst,
                                 std::vector<vector<double>> &input, vector<double> &real,
                                 std::vector<double> &result) {
//    int N = input.size();
    result.resize(N, 0.);

    double fitness = 0.;
//    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    for (int i = 0; i < N; i++) {
        result[i] = h_evaluateIndividual(program, programConst, input[i], program.size());
        fitness += fabs(real[i] - result[i]);
    }
//    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
//    std::cerr << "CPU Time difference [us] = "
//              << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << std::endl;
    return fitness;
}

double CudaEvaluator::h_evaluateNew(char* postfixMem, uint PROG_SIZE, uint MEM_SIZE, std::vector<double> &result) {
//    int N = input.size();
    result.resize(N, 0.);

    double fitness = 0.;
//    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    for (int i = 0; i < N; i++) {
        result[i] = h_evaluateIndividualNew(postfixMem, PROG_SIZE, MEM_SIZE, datasetInput[i]);
        fitness += fabs(datasetOutput[i] - result[i]);
    }
//    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
//    std::cerr << "CPU Time difference [us] = "
//              << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << std::endl;
    return fitness;
}

/*
void CudaEvaluator::evaluate(vector<uint> &postfix, vector<double> &postfixConstants) {

    // evaluiraj na cpu
    vector<double> h_result;
    double h_fitness = h_evaluate(postfix, postfixConstants, datasetInput, datasetOutput, h_result);

    // evaluiraj na gpu
    vector<double> d_result;
    double d_fitness = d_evaluate(postfix, postfixConstants, datasetInput, datasetOutput, d_result);

    // provjeri jesu li jednaki
//    for (int i = 0; i < h_result.size(); i++) {
//        if (fabs(h_result[i] - d_result[i]) > 1E-10) {     // std::numeric_limits<double>::epsilon()
//            cerr << "FAIL\t" << "host:\t" << h_result[i] << "\tdev:\t" << d_result[i] << endl;
//        }
//    }

    cerr << "host:\t" << h_fitness << "\tdev:\t" << d_fitness << endl;

}
 */